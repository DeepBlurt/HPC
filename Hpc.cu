#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include"matrix.h"
#include<cstdlib>
#include<cmath>
#include"layer.h"
#include<iostream>

// Matrix multiplication kernel called by MatMul()
__device__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < A.height && col < B.width)
    {
    	for(int e = 0; e < A.width; ++e)
    		Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
    	C.elements[row * C.width + col] = Cvalue;
    }
}



__device__ void AddSigmoidKernel(Matrix A, Matrix B, Matrix C)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row < A.height  && col < A.width)
	{
		C.elements[row * C.width + col] = A.elements[row * A.width + col]\
				+ B.elements[row * B.width + col];
		C.elements[row * C.width + col] = \
				1.0 / (1.0 + std::exp(-C.elements[row * C.width + col]));
	}
}



__global__ void forwardKernel(Matrix input, Matrix weight, Matrix bias, \
		Matrix output)
{
	MatMulKernel(input, weight, output);
	__syncthreads();
	AddSigmoidKernel(output, bias, output);
	__syncthreads();
}

extern "C"
int deviceQuery()
{
    // By default, we use device 0, otherwise
	//we override the device ID based on what is provided at the command line
    int devID = 0;

    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);

    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    else
    {
        //printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    // Use a larger block size for Fermi and above
    int block_size = (deviceProp.major < 2) ? 16 : 32;
    return block_size;
}


extern "C"
void forwardGpu(Matrix input, Net* n, Matrix output)
{
	//std::cout<<"in forwardGPU...\n";
	hipError_t err;
	// copy the net parameter
	// compute dimGrid, dimBlock
	int block_size = deviceQuery();
	dim3 dimBlock(block_size, block_size);
	//std::cout<<block_size<<" ";
	dim3 dimGrid(4096 / dimBlock.x, 4096 / dimBlock.y);
	//device memory....
	Matrix h_input;
	h_input.width = 1;
	h_input.height = input.height;
	h_input.elements = (float*)malloc(input.height*sizeof(float));
	for(int i = 0; i < input.height; ++i)
		h_input.elements[i] = input.elements[i];

	for(int i = 0; i < n->numLayer; ++i)
	{
		// allocate bias
		int outDim = n->layerOutputDim[i];
		Matrix d_bias;
		d_bias.width = 1;
		d_bias.height = outDim;
		d_bias.elements = NULL;
		size_t size = d_bias.height * sizeof(float);
		err = hipMalloc((void**)&d_bias.elements, size);
		if(err != hipSuccess)
		{
			std::cout<<"Malloc failed...\n";
		}
		hipMemcpy(d_bias.elements, n->layers[i].bias.elements, size,\
						hipMemcpyHostToDevice);
		// allocate weights
		Matrix d_weight;
		d_weight.width = n->layerOutputDim[i];
		d_weight.height = n->layerInputDim[i];
		// copy weight parameters
		size = d_weight.width * d_weight.height * sizeof(float);

		err = hipMalloc(&d_weight.elements, size);
		if(err !=hipSuccess)
			std::cout<<"Malloc failed...\n";
		hipMemcpy(d_weight.elements, n->layers[i].weight.elements, \
						size, hipMemcpyHostToDevice);
		// allocate output
		Matrix d_outTemp;
		d_outTemp.width = 1;
		d_outTemp.height = n->layerOutputDim[i];
		size = n->layerOutputDim[i] * sizeof(float);
		err = hipMalloc(&d_outTemp.elements, size);
		if(err != hipSuccess)
			std::cout<<"Malloc d_outtemp failed\n";

		// allocate inputs
		Matrix d_input;
		d_input.height = n->layerInputDim[i];
		d_input.width = 1;
		size = d_input.height * sizeof(float);
		err = hipMalloc(&d_input.elements, size);
		if(err != hipSuccess)
			std::cout<<"cudamalloc d_input failed";
		hipMemcpy(d_input.elements, h_input.elements, \
				size, hipMemcpyHostToDevice);
		// Invoke kernel calls
		forwardKernel<<<dimGrid, dimBlock>>>(d_input, d_weight, d_bias,\
				d_outTemp);
		//std::cout<<"after kernel invoked\n";
		// copy current output;
		Matrix h_output;
		h_output.width = 1;
		h_output.height = n->layerOutputDim[i];
		h_output.elements = new float[h_output.height];
		size = h_output.height * sizeof(float);
		hipMemcpy(h_output.elements, d_outTemp.elements, size, hipMemcpyDeviceToHost);
		// copy h_output to h_input;
		delete h_input.elements;
		h_input.width = 1;
		h_input.height = h_output.height;
		h_input.elements = new float[h_output.height];
		for(int j = 0; j < h_output.height; ++j)
			h_input.elements[i] = h_output.elements[i];
		// free memory..
		if(i == n->numLayer -1 )
		{
			for(int k = 0; k < h_output.height; ++k)
				output.elements[k] = h_output.elements[k];
		}
		delete h_output.elements;
		hipFree(d_input.elements);
		hipFree(d_outTemp.elements);
		hipFree(d_bias.elements);
		hipFree(d_weight.elements);
	}
	// copy output to return
}

